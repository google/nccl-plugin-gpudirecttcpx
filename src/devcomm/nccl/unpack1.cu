/*
 Copyright 2024 Google LLC

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

      https://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

#include "unpack1.h"

#include "hip/hip_runtime.h"

#include "checks1.h"
#include "cuda_checks.h"
#include "cuda_wrapper.h"
#include "debug1.h"

#ifdef DRIVER_API

tcpxResult_t __tcpxNetDeviceQueueNew(void* gpu_dev, bool passive, void** handle, void** d_handle) {
  TCPXCHECK(gpu_push_current(gpu_dev));

  struct tcpxNetDeviceQueue* h;
  struct unpackNetDeviceHandle* d;
  INFO(TCPX_NET, "NetDeviceHandle size %zu", sizeof *h);
  INFO(TCPX_NET, "NetDeviceDevHandle size %zu", sizeof *d);

  // clang-format off

  // host side handle
  CUASSERT(hipHostAlloc((void**) &h, sizeof *h, 0));
  memset(h, 0, sizeof *h);
  CUASSERT(hipHostAlloc((void**) &(h->meta), sizeof *(h->meta), 
                              hipHostMallocMapped
                            | hipHostMallocPortable));
                            // | hipHostMallocWriteCombined));
  h->gpu_dev = gpu_dev;
  h->head = h->tail = 0;

  INFO(TCPX_NET, "handle %p size %zu", h, sizeof *h);
  INFO(TCPX_NET, "h->meta %p size %zu", h->meta, sizeof *(h->meta));

  // cuda side handle
  CUASSERT(hipMalloc((hipDeviceptr_t*) &d, sizeof *d));
  struct unpackNetDeviceHandle h_d;
  CUASSERT(hipHostGetDevicePointer((hipDeviceptr_t*) &(h_d.meta), h->meta, 0));

  if (passive) {
    TCPXASSERT(gpu_get_rxmem(gpu_dev, &(h_d.bounce_buf)));
  }

  // initialize nccl side head, nccl side increments the counter prior to performing copy
  h_d.head = (uint64_t) -1;

  CUASSERT(hipMemcpyHtoD((hipDeviceptr_t) d, &h_d, sizeof h_d));

  TCPXCHECK(gpu_pop_current(nullptr, nullptr));  // we don't care about output

  *handle = h;
  *d_handle = d;

  return tcpxSuccess;
}

void __tcpxNetDeviceQueueFree(void* handle, void* d_handle) {
  struct tcpxNetDeviceQueue* h =
      static_cast<struct tcpxNetDeviceQueue*>(handle);
  struct unpackNetDeviceHandle* d =
      static_cast<struct unpackNetDeviceHandle*>(d_handle);

  TCPXASSERT(gpu_push_current(h->gpu_dev));

  CUASSERT(hipFree((hipDeviceptr_t) d));
  CUASSERT(hipHostFree(h->meta));
  CUASSERT(hipHostFree(h));

  TCPXASSERT(gpu_pop_current(nullptr, nullptr));  // we don't care about output
}

#else // not DRIVER_API

tcpxResult_t __tcpxNetDeviceQueueNew(void* gpu_dev, bool passive, void** handle, void** d_handle) {
  TCPXCHECK(gpu_push_current(gpu_dev));

  struct tcpxNetDeviceQueue* h;
  struct unpackNetDeviceHandle* d;
  INFO(NCCL_NET, "NetDeviceHandle size %zu", sizeof *h);
  INFO(NCCL_NET, "NetDeviceDevHandle size %zu", sizeof *d);

  // clang-format off

  // host side handle
  CUDACHECK(hipHostAlloc((void**) &h, sizeof *h, 0));
  memset(h, 0, sizeof *h);
  CUDACHECK(hipHostAlloc((void**) &(h->meta), sizeof *(h->meta), 
                              hipHostMallocMapped
                            | hipHostMallocPortable));
  h->gpu_dev = gpu_dev;
  h->head = h->tail = 0;

  INFO(NCCL_NET, "handle %p size %zu", h, sizeof *h);
  INFO(NCCL_NET, "h->meta %p size %zu", h->meta, sizeof *(h->meta));

  // cuda side handle
  CUDACHECK(hipMalloc(&d, sizeof *d));
  struct unpackNetDeviceHandle h_d;
  CUDACHECK(hipHostGetDevicePointer(&(h_d.meta), h->meta, 0));
  if (passive) {
    NCCLASSERT(gpu_get_rxmem(gpu_dev, &(h_d.bounce_buf)));
  }
  // INFO(NCCL_NET, "111 setting meta to %p [%p,%p], bounce_buf to %p", h_d.meta, h_d.meta->mem, h_d.meta->cnt, h_d.bounce_buf);
  CUDACHECK(hipMemcpy(d, &h_d, sizeof h_d, hipMemcpyHostToDevice));

  *handle = h;
  *d_handle = d;

  return tcpxSuccess;
}

void __tcpxNetDeviceQueueFree(void* handle, void* d_handle) {
  struct tcpxNetDeviceQueue* h =
      static_cast<struct tcpxNetDeviceQueue*>(handle);
  struct unpackNetDeviceHandle* d =
      static_cast<struct unpackNetDeviceHandle*>(d_handle);

  TCPXASSERT(gpu_push_current(h->gpu_dev));

  CUDAASSERT(hipFree(d));
  CUDAASSERT(hipHostFree(h->meta));
  CUDAASSERT(hipHostFree(h));
}

#endif
