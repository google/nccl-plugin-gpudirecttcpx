/*
 Copyright 2024 Google LLC

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

      https://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

#include "inline.h"

#include "hip/hip_runtime.h"

#include "alloc1.h"
#include "checks1.h"
#include "cuda_wrapper.h"
#include "cuda_checks.h"
#include "debug1.h"

struct inlineHandle {
  void* gpu_dev;
};

tcpxResult_t __gpu_inline_alloc(void* gpu_dev, void** inline_handle) {
  struct inlineHandle* _inline_handle;
  TCPXCHECK(tcpxCalloc(&_inline_handle, 1));
  _inline_handle->gpu_dev = gpu_dev;

  *inline_handle = _inline_handle;
  return tcpxSuccess;
}

tcpxResult_t __gpu_inline_free(void* inline_handle) {
  struct inlineHandle* _inline_handle = (struct inlineHandle*) inline_handle;

  free(_inline_handle);

  return tcpxSuccess;
}

tcpxResult_t __gpu_inline_memcpy(void* inline_handle, void* dst, void* src, size_t len) {
  return tcpxSuccess;
}

tcpxResult_t __gpu_inline_sync(void* inline_handle) {
  return tcpxSuccess;
}

tcpxResult_t __gpu_inline_test(void* inline_handle, int* done) {
  *done = 1;

  return tcpxSuccess;
}
