/*
 Copyright 2024 Google LLC

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

      https://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

#include "rx_pool.h"

#include <errno.h>
#include <error.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "debug1.h"
#include "flags.h"
#include "unix_socket_client.h"
#include "cuda_checks.h"

hipDeviceptr_t gpumem_import(char* gpu_pci_addr) {
  // std::string nvdmad_path = absl::StrFormat("/tmp/nvdma-%s", gpu_pci_addr);
  char nvdmad_path[128];
  sprintf(nvdmad_path, "/tmp/nvdma-%s", gpu_pci_addr);
  hipIpcMemHandle_t mem_handle;
  hipDeviceptr_t ptr;
  int fd, ret;

  fd = open(nvdmad_path, O_RDONLY);
  if (fd == -1) {
    INFO(TCPX_NET, "skip %s", nvdmad_path);
    return 0;
    // error(1, errno, "open %s", nvdmad_path.c_str());
  }

  ret = read(fd, &mem_handle, sizeof(mem_handle));
  if (ret == -1) error(1, errno, "read %s", nvdmad_path);
  if (ret != sizeof(mem_handle))
    error(1, 0, "read %s: %dB", nvdmad_path, ret);
  if (close(fd)) error(1, errno, "close %s", nvdmad_path);

  CUASSERT(
      hipIpcOpenMemHandle(&ptr, mem_handle, hipIpcMemLazyEnablePeerAccess));
  return ptr;
}

struct IpcGpuMemFdMetadata {
  int fd{-1};
  size_t size{0};
  size_t align{0};
};
CuIpcMemfdHandle* GpumemImport(const hipCtx_t ctx, const char* gpu_pci_addr, const char* ipc_prefix) {
  char client_id[128];
  IpcGpuMemFdMetadata gpumem_fd_metadata;

  // fetch ipc shareable fd
  snprintf(client_id, 128, "%s/get_gpu_fd_%s", ipc_prefix, gpu_pci_addr);
  UnixSocketClient gpumem_fd_by_gpu_pci_client(client_id);
  absl::Status status = gpumem_fd_by_gpu_pci_client.Connect();
  if (!status.ok()) {
    WARN("unix client %s, gpumem fd client connect failed %s", client_id, std::string(status.message()).c_str());
    return nullptr;
  }
  UnixSocketMessage req;
  UnixSocketProto* req_mutable_proto = req.mutable_proto();
  req_mutable_proto->set_raw_bytes(gpu_pci_addr);
  gpumem_fd_by_gpu_pci_client.Send(req);
  absl::StatusOr<UnixSocketMessage> resp =
      gpumem_fd_by_gpu_pci_client.Receive();
  if (!resp.status().ok()) {
    WARN("unix client %s, recv fd failed %s", client_id, std::string(resp.status().message()).c_str());
    return nullptr;
  }
  if (!resp.value().has_fd() || resp.value().fd() < 0) {
    WARN("unix client %s, GPU fd not found %s", client_id, gpu_pci_addr);
    return nullptr;
  }

  // fetch gpu memory metadata
  snprintf(client_id, 128, "%s/get_gpu_metadata_%s", ipc_prefix, gpu_pci_addr);
  UnixSocketClient gpumem_metadata_by_gpu_pci_client(client_id);
  absl::Status status1 = gpumem_metadata_by_gpu_pci_client.Connect();
  if (!status1.ok()) {
    WARN("unix client %s, gpumem metadata client connect failed %s", client_id, std::string(status1.message()).c_str());
    return nullptr;
  }
  UnixSocketMessage req_metadata;
  UnixSocketProto* md_mutable_proto = req_metadata.mutable_proto();
  md_mutable_proto->set_raw_bytes(gpu_pci_addr);
  gpumem_metadata_by_gpu_pci_client.Send(req_metadata);
  absl::StatusOr<UnixSocketMessage> resp_metadata =
      gpumem_metadata_by_gpu_pci_client.Receive();
  if (!resp_metadata.status().ok()) {
    WARN("unix client %s, recv metadata failed %s", client_id, std::string(resp_metadata.status().message()).c_str());
    return nullptr;
  }
  if (!resp_metadata.value().has_proto() ||
      !resp_metadata.value().proto().has_raw_bytes()) {
    WARN("unix client %s, GPU metadata not found %s", client_id, gpu_pci_addr);
    return nullptr;
  } else {
    memcpy((void*)&gpumem_fd_metadata,
           (void*)resp_metadata.value().proto().raw_bytes().data(),
           resp_metadata.value().proto().raw_bytes().size());
  }
  int dev_id;
  CUDAASSERT(hipDeviceGetByPCIBusId(&dev_id, gpu_pci_addr));
  return new CuIpcMemfdHandle(ctx, resp.value().fd(), dev_id,
                              gpumem_fd_metadata.size,
                              gpumem_fd_metadata.align);
}
CuIpcMemfdHandle* GpumemImport(const hipCtx_t ctx, const char* gpu_pci_addr) {
  return GpumemImport(ctx, gpu_pci_addr, kUnixClientPrefix);
}
