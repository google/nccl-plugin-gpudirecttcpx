/*
 Copyright 2024 Google LLC

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

      https://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

#include "common.h"

#include <unistd.h>

#include "cuda_checks.h"
#include "debug1.h"
#include "ret.h"

CuIpcMemfdHandle::CuIpcMemfdHandle(hipCtx_t ctx, int fd, int dev_id, size_t size, size_t align) {
  ctx_ = ctx;
  CUASSERT(hipCtxPushCurrent(ctx_));
  hipCtxGetDevice(&dev_);
  if ((int) dev_ != dev_id) {
    WARN("unexpected dev_id %d, vs input %d", (int) dev_, dev_id);
    dev_id = (int) dev_;
  }

  INFO(TCPX_NET, "Importing CUDA IPC mem from from fd: %ld, dev_id: %ld, size: %ld, "
       "align: %ld", fd, dev_id, size, align);
  // CUASSERT(hipDeviceGet(&dev_, dev_id));
  // CUASSERT(hipDevicePrimaryCtxRetain(&ctx_, dev_));

  size_ = size;
  CUASSERT(
      hipMemImportFromShareableHandle(
        &handle_, (void*)(long long)fd,
        hipMemHandleTypePosixFileDescriptor));
  CUASSERT(hipMemAddressReserve(&ptr_, size_, align, 0, 0));
  CUASSERT(hipMemMap(ptr_, size_, 0, handle_, 0));
  close(fd);
  hipMemAccessDesc desc = {};
  desc.location.type = hipMemLocationTypeDevice;
  desc.location.id = dev_id;
  desc.flags = hipMemAccessFlagsProtReadWrite;
  CUASSERT(hipMemSetAccess(ptr_, size_, &desc, 1 /*count*/));

  hipCtx_t _;
  CUASSERT(hipCtxPopCurrent(&_));
}
CuIpcMemfdHandle::~CuIpcMemfdHandle() {
  CUASSERT(hipCtxPushCurrent(ctx_));

  hipMemUnmap(ptr_, size_);
  hipMemRelease(handle_);
  hipMemAddressFree(ptr_, size_);

  hipCtx_t _;
  CUASSERT(hipCtxPopCurrent(&_));
}
