/*
 Copyright 2024 Google LLC

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

      https://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

#include "cuda_wrapper.h"

#include <ctype.h>
#include <memory>
#include <unistd.h>

#ifdef DRIVER_API
#include "hip/hip_runtime.h"
#else
#include "hip/hip_runtime.h"
#endif

#include "alloc1.h"
#include "checks1.h"
#include "common.h"
#include "cuda_checks.h"
#include "param1.h"
#include "rx_pool.h"
#include "tx_pool.h"
#include "../flags.h"
#include "../macro.h"

bool init = 0;
void lazyInit() {
#ifdef DRIVER_API
  if (!init)
    CUASSERT(hipInit(0));
#else
#endif
  init = 1; 
}

tcpxResult_t gpu_n_dev(int* n) {
  lazyInit();

  int _n;
#ifdef DRIVER_API
  CUASSERT(hipGetDeviceCount(&_n));
#else
  CUDAASSERT(hipGetDeviceCount(&_n));
#endif
  if (_n <= 0) {
    WARN("NET/" PRODUCT_NAME "/CUDA : no cuDevices found");
    return tcpxInternalError;
  }

  INFO(TCPX_NET | TCPX_INIT, "NET/" PRODUCT_NAME "/CUDA : cuDevices count %d", _n);
  *n = _n;
  return tcpxSuccess;
}

tcpxResult_t gpu_init_internal(struct gpuDev *gpu, int ordinal) {
  lazyInit();

  int i = ordinal;

  char *pci_addr = gpu->pci_addr;

#ifdef DRIVER_API
  INFO(NCCL_NET, "new cuda context on dev %d", i);
  CUASSERT(hipDeviceGet(&(gpu->dev), i));

  if (kCudaUsePrimaryCtx) {
    CUASSERT(hipDevicePrimaryCtxRetain(&(gpu->ctx), gpu->dev));
  } else {
    CUASSERT(hipCtxCreate(&(gpu->ctx), hipDeviceMapHost, gpu->dev));
    hipCtx_t _;
    CUASSERT(hipCtxPopCurrent(&_));
  }

  CUASSERT(hipDeviceGetPCIBusId(pci_addr, CU_PCI_ADDR_LEN, gpu->dev));
#else
  CUDACHECK(hipDeviceGetPCIBusId(pci_addr, CU_PCI_ADDR_LEN, gpu->dev));
#endif
  gpu->dev = i;
  for (int j = 0; j < CU_PCI_ADDR_LEN; j++) {
    pci_addr[j] = tolower(pci_addr[j]);
  }
  INFO(TCPX_NET, "NET/" PRODUCT_NAME "/CUDA : gpu dev %d pci [%s](%zu)", i, pci_addr, strlen(pci_addr));

  return tcpxSuccess;
}

TCPX_PARAM(UseGpuPciClient, "TCPX_RXMEM_IMPORT_USE_GPU_PCI_CLIENT", 1);
TCPX_PARAM(RxmemSkipOdds, "TCPX_RXMEM_IMPORT_SKIP_ODDS", 0);

struct gpuDev* _gpus = nullptr;
int _n;

tcpxResult_t gpu_init(void** gpus, int n) {
  if (_gpus != nullptr) {
    WARN("gpu lib init twice");
    return ncclInternalError;
  }
  TCPXCHECK(tcpxCalloc(&_gpus, n));

  for (int i = 0; i < n; i++) {
#ifdef GPU_LAZY_INIT
    _gpus[i].dev = -1;
#else
    gpu_init_internal(_gpus + i, i);
#endif
  }

  *gpus = _gpus;
  _n = n;
  return tcpxSuccess;
}

tcpxResult_t gpu_deinit(void* gpus) {
  free(gpus);
  return tcpxSuccess;
}

// a bit hacky requires setting primary context first by application
tcpxResult_t gpu_current_dev(void *gpus, void **gpu) {
  struct gpuDev *_gpus = (struct gpuDev*) gpus;
  int ordinal;
  CUDACHECK(hipGetDevice(&ordinal));
  *gpu = _gpus + ordinal;
#ifdef GPU_LAZY_INIT
  if ((_gpus + ordinal)->dev < 0) {
    gpu_init_internal(_gpus + ordinal, ordinal);
  }
#endif
  return tcpxSuccess;
}

tcpxResult_t gpu_push_current(void* gpu) {
  struct gpuDev *_gpu = (struct gpuDev*) gpu;
  int ordinal = _gpu - _gpus;
  if (ordinal >= _n) {
    WARN("invalid GPU %d/%d", ordinal, _n);
    return ncclInternalError;
  }
#ifdef GPU_LAZY_INIT
  if (_gpu->dev < 0) {
    gpu_init_internal(_gpu, ordinal);
  }
#endif
#ifdef DRIVER_API
  CUCHECK(hipCtxPushCurrent(_gpu->ctx));
#else
  CUDACHECK(hipSetDevice(_gpu->dev));
#endif
  return tcpxSuccess;
}

tcpxResult_t gpu_pop_current(void* gpus, void** gpu) {
  if (gpus && gpu) {
    TCPXCHECK(gpu_current_dev(gpus, gpu));
  }
#ifdef DRIVER_API
  hipCtx_t ctx;
  CUCHECK(hipCtxPopCurrent(&ctx));
#else
#endif
  return tcpxSuccess;
}

// lazy init
tcpxResult_t gpu_rx_init_internal(struct gpuDev* gpu) {
  bool use_gpu_pci_client = TCPX_GET_PARAM(UseGpuPciClient);
  bool rx_8_gpu = !TCPX_GET_PARAM(RxmemSkipOdds);

  char *gpu_pci_addr = gpu->pci_addr;
  int ordinal;
  TCPXCHECK(gpu_ordinal(gpu, &ordinal));

  if (!use_gpu_pci_client) {
#ifdef DRIVER_API
    gpu->rx.rxmem = gpumem_import(gpu_pci_addr);
#else
    gpu->rx.rxmem = (void *)gpumem_import(gpu_pci_addr);
#endif
  } else {

    if (!rx_8_gpu && ordinal % 2 == 1) {
      gpu->rx.handle = nullptr;
      gpu->rx.rxmem = 0;
    } else {
      gpu->rx.handle =
          std::unique_ptr<CuIpcMemfdHandle>(GpumemImport(gpu->ctx, gpu_pci_addr));
      if (!gpu->rx.handle) {
        return tcpxSystemError;
      }
#ifdef DRIVER_API
      gpu->rx.rxmem = gpu->rx.handle->GetGpuMem();
#else
      gpu->rx.rxmem = (void *)gpu->rx.handle->GetGpuMem();
#endif
    }
  }

  return tcpxSuccess;
}

tcpxResult_t gpu_get_rxmem(void* gpu, void** rxmem) {
  struct gpuDev *_gpu = (struct gpuDev *)gpu;
  if ((void*) _gpu->rx.rxmem == nullptr) {
    TCPXCHECK(gpu_rx_init_internal(_gpu));
  }
  *rxmem = (void*) _gpu->rx.rxmem;
  char buf[CU_PCI_ADDR_LEN];
  gpu_pci_addr(gpu, buf, CU_PCI_ADDR_LEN);
  INFO(TCPX_NET, "gpu [%s] rxmem %s", buf, *rxmem ? "ready" : "skip");
  return tcpxSuccess;
}

tcpxResult_t gpu_tx_reg_mr(void* gpu, void** gpu_tx, int* fd, char* nic_pci_addr, void* buf, size_t sz) {
  struct gpuDev *_gpu = (struct gpuDev *)gpu;

  struct gpuTx *_gpu_tx;

  TCPXCHECK(gpu_push_current(gpu));

  TCPXCHECK(tcpxCalloc(&_gpu_tx, 1));
  int ret = get_gpumem_dmabuf_pages_fd(_gpu->pci_addr, nic_pci_addr,
                                 (hipDeviceptr_t)buf, sz, &(_gpu_tx->dma_buf_fd));
  if (ret < 0) {
    WARN("gpu_tx_reg_mr failed %d", ret);
    return tcpxInternalError;
  }
  _gpu_tx->gpu_mem_fd = ret;

  TCPXCHECK(gpu_pop_current(nullptr, nullptr));

  *gpu_tx = _gpu_tx;
  *fd = _gpu_tx->gpu_mem_fd;
  return tcpxSuccess;
}

tcpxResult_t gpu_tx_dereg_mr(void* gpu, void* gpu_tx) {
  struct gpuTx *_gpu_tx = (struct gpuTx *)gpu_tx;

  if (_gpu_tx->dma_buf_fd) close(_gpu_tx->dma_buf_fd);

  return tcpxSuccess;
}

tcpxResult_t gpu_pci_addr(void* gpu, char *buf, int len) {
  struct gpuDev *_gpu = (struct gpuDev *)gpu;

  if (len < CU_PCI_ADDR_LEN)  {
    WARN("gpu_pci_addr buf too short %d vs expected at least %d", len, CU_PCI_ADDR_LEN);
  }

  memcpy(buf, _gpu->pci_addr, CU_PCI_ADDR_LEN);

  return tcpxSuccess;
}

char *gpu_tostring(void* gpu, char *buf, int n) {
  if (gpu == nullptr) snprintf(buf, n, "null");

  struct gpuDev *_gpu = (struct gpuDev *)gpu;

  snprintf(buf, n, "cudaDev %d", _gpu->dev);
  return buf;
}

tcpxResult_t gpu_node(void* gpu, int *n) {
  struct gpuDev* _gpu = (struct gpuDev*) gpu;
  *n = _gpu->dev < 4 ? 0 : 1;
  return tcpxSuccess;
}

tcpxResult_t gpu_ordinal(void* gpu, /*output=*/int *ordinal) {
  struct gpuDev* _gpu = (struct gpuDev*) gpu;
  *ordinal = _gpu->dev;
  return tcpxSuccess;
}
