/*
 Copyright 2024 Google LLC

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

      https://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

#include "tx_pool.h"

#include <errno.h>
#include <fcntl.h>
#include <stdio.h>
#include <string.h>
#include <sys/ioctl.h>
#include <unistd.h>

#include "common.h"
#include "debug1.h"
#include "param1.h"

#include "../macro.h"


/**** p2pdma begin ****/

const char cfg_procfs_prefix[] = "/proc/driver/nvdma";

struct page_vec_create_info {
  unsigned long gpu_vaddr;
  unsigned long size;
};

#define PAGE_VEC_CREATE _IOW('c', 'c', struct page_vec_create_info)
#define PAGE_VEC_START_OFF _IOR('c', 'o', int)

int get_gpumem_pages_fd(char pci_addr[16], hipDeviceptr_t gpu_mem,
                         size_t gpu_mem_sz, int* align_offp) {
  char path[256];
  sprintf(path, "%s/%s/new_fd", cfg_procfs_prefix, pci_addr);

  int ret;
  int fd = open(path, O_WRONLY);
  if (fd == -1) {
    WARN("Error opening %s", path);
    return -EBADF;
  }

  struct page_vec_create_info create_info = {(unsigned long) gpu_mem, gpu_mem_sz};
  ret = ioctl(fd, PAGE_VEC_CREATE, &create_info);
  if (ret < 0) {
    WARN("ioctl() failed: %s", strerror(errno));
    goto err_close;
  }

  if (close(fd)) {
    WARN("close: %s", strerror(errno));
    return -EIO;
  }

  INFO(TCPX_NET, "Registered region 0x%lx of %lu Bytes", gpu_mem, gpu_mem_sz);
  if (ioctl(ret, PAGE_VEC_START_OFF, align_offp)) {
    WARN("Error getting start offset: %s", strerror(errno));
  }
  return ret;

err_close:
  close(fd);
  return -EIO;
}

/**** p2pdma end ****/

/**** dmabuf begin ****/

#include "linux/types.h"

// internal ioctl API structs
const char cfg_nvp2pdmabuf_procfs_prefix[] = "/proc/driver/nvp2p_dma_buf";
struct gpumem_dma_buf_create_info {
  unsigned long gpu_vaddr;
  unsigned long size;
};
#define GPUMEM_DMA_BUF_CREATE _IOW('c', 'c', struct gpumem_dma_buf_create_info)

struct dma_buf_create_pages_info {
  __u64 pci_bdf[3];
  __s32 dma_buf_fd;
  __s32 create_page_pool;
};

#define DMA_BUF_BASE 'b'
#define DMA_BUF_CREATE_PAGES _IOW(DMA_BUF_BASE, 2, struct dma_buf_create_pages_info)
TCPX_PARAM(RegDmabufUseInternalApi, "GPUDIRECTTCPX_REG_DMABUF_USE_INTERNAL_API", 0);

int get_gpumem_dmabuf_pages_fd(char* gpu_pci_addr, char* nic_pci_addr, hipDeviceptr_t gpu_mem, size_t gpu_mem_sz, int* dma_buf_fd) {
  int err, ret, fd;

  if (TCPX_GET_PARAM(RegDmabufUseInternalApi) == 0) {
    hipError_t cu_ret;
    cu_ret = hipMemGetHandleForAddressRange((void*)dma_buf_fd, (hipDeviceptr_t)gpu_mem,
                                  gpu_mem_sz, hipMemRangeHandleTypeDmaBufFd,
                                  0);

    if (*dma_buf_fd < 0) {
      const char* name, *msg;
      hipDrvGetErrorName(cu_ret, &name);
      hipDrvGetErrorString(cu_ret, &msg);
      WARN("hipMemGetHandleForAddressRange (%p, %zu) failed, %s %s", gpu_mem, gpu_mem_sz, name, msg);
      return -1;
    }
  } else
  {
    char path[256];
    sprintf(path, "%s/%s/new_fd", cfg_nvp2pdmabuf_procfs_prefix, gpu_pci_addr);

    fd = open(path, O_WRONLY);
    if (fd == -1) {
      WARN("Error opening %s", path);
      return -EBADF;
    }

    INFO(TCPX_NET, "create_info = { %p, %zu }", gpu_mem, gpu_mem_sz);
    struct gpumem_dma_buf_create_info create_info = {gpu_mem, gpu_mem_sz};
    ret = ioctl(fd, GPUMEM_DMA_BUF_CREATE, &create_info);
    if (ret < 0) {
      perror("ioctl gpumem dma_buf create");
      err = -EIO;
      goto err_close;
    }

    if (close(fd)) {
      perror("close");
      err = -EIO;
      return err;
    }

    *dma_buf_fd = ret;
  }

  INFO(TCPX_INIT | TCPX_NET,
       "NET/" PRODUCT_NAME ": Registered dmabuf region 0x%lx of %lu Bytes", gpu_mem,
       gpu_mem_sz);
  struct dma_buf_create_pages_info info;
  info.dma_buf_fd = *dma_buf_fd;
  info.create_page_pool = 0;

  uint16_t pci_bdf[3];
  ret = sscanf(nic_pci_addr, "0000:%hx:%hx.%hx",
               &pci_bdf[0], &pci_bdf[1], &pci_bdf[2]);
  info.pci_bdf[0] = pci_bdf[0];
  info.pci_bdf[1] = pci_bdf[1];
  info.pci_bdf[2] = pci_bdf[2];
  if (ret != 3) {
    err = -EINVAL;
    goto err_close_dmabuf;
  }

  ret = ioctl(*dma_buf_fd, DMA_BUF_CREATE_PAGES, &info);
  if (ret < 0) {
    perror("ioctl get dma_buf frags");
    err = -EIO;
    goto err_close_dmabuf;
  }
  return ret;

err_close_dmabuf:
  close(*dma_buf_fd);
  return err;
err_close:
  close(fd);
  return err;
}

/**** dmabuf end ****/